#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2019 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdint.h>
#include <stdio.h>
#include "common.h"
#include "utils/utils.h"
#include "utils/channel.hpp"

extern "C" __device__ __noinline__ void instrument_mem(int pred, int opcode_id,
                                                       uint64_t addr,
                                                       uint64_t grid_launch_id,
                                                       uint64_t pchannel_dev) {
    char formatted_msg[] = "your formtted message\n";
    /* if thread is predicated off, return */
    if (!pred) {
        return;
    }

    // formatted_msg = your_sprintf("opcode: %d, addr %x\n", opcode_id, addr);
    ((ChannelDev*)pchannel_dev)->push(formatted_msg, sizeof(formatted_msg));
}

extern "C" __device__ __noinline__ void instrument_call(int pred, int opcode_id,
                                                       uint64_t addr,
                                                       uint64_t grid_launch_id,
                                                       uint64_t pchannel_dev) {
    char formatted_msg[] = "your formtted message\n";
    //char formatted_msg[100];
    /* if thread is predicated off, return */
    if (!pred) {
        return;
    }
    call_trace_t ct;
    ct.opcode_id = opcode_id;
    int active_mask = __ballot_sync(__activemask(), 1);
    const int laneid = get_laneid();
    const int first_laneid = __ffs(active_mask) - 1;

    /* collect memory address information from other threads */
    //for (int i = 0; i < 32; i++) {
    //    ct.addrs[i] = __shfl_sync(active_mask, addr, i);
    //}
    ct.grid_launch_id = grid_launch_id;
    ct.cta_id_x = get_ctaid().x;
    ct.cta_id_y = get_ctaid().y;
    ct.cta_id_z = get_ctaid().z;
    ct.warp_id = get_warpid();
    int character_limit = 2048;
    int current_character = 0;
    while(character_limit > current_character)
    {
        unsigned long instruction = *((unsigned long *)addr+current_character*1);
        ct.call_sass[current_character] = instruction;
        current_character += 1;
    }
    //ct.call_sass[0] = (unsigned long) *((unsigned long*)addr);
    //ct.call_sass[0] = *((unsigned long*) 0x700b6f400);

    //while(character_limit > current_character)
    //{
    //    unsigned long instruction = *((unsigned long *)ct.addrs[0]+current_character*16);
    //    ct.call_sass[current_character] = instruction;
    //    current_character += 1;
    //}

    // formatted_msg = your_sprintf("opcode: %d, addr %x\n", opcode_id, addr);
    ((ChannelDev*)pchannel_dev)->push(&ct, sizeof(call_trace_t));
}
